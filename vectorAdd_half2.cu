#include "hip/hip_runtime.h"

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "hip/hip_complex.h"
#include "hip/hip_fp16.h"
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
struct cmplx16 {
    __half x;
    __half y;
};

__global__ void
vectorAdd(const cmplx16 *A, const cmplx16 *B, cmplx16 *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
    #if __CUDA_ARCH__ >= 530
            C[i].x = __hadd(A[i].x, B[i].x);
            C[i].y = __hadd(A[i].y, B[i].y);
    #else
       C[i].x = __float2half(__half2float(A[i].x) + __half2float(B[i].x));
        C[i].y = __float2half(__half2float(A[i].y) + __half2float(B[i].y));
    #endif
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(cmplx16);
    printf("[Vector addition of %d elements]\n", numElements);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the host input vector A
    cmplx16 *h_A = (cmplx16*)malloc(size);

    // Allocate the host input vector B
    cmplx16 *h_B = (cmplx16*)malloc(size);

    // Allocate the host output vector C
    cmplx16 *h_C = (cmplx16*)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }


        
    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i].x = rand() / (float)RAND_MAX;
        h_A[i].y = rand() / (float)RAND_MAX;
        h_B[i].x = rand() / (float)RAND_MAX;
        h_B[i].y = rand() / (float)RAND_MAX;


    }

    // Allocate the device input vector A
    cmplx16 *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    cmplx16 *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    cmplx16 *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    hipEventRecord(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipEventRecord(stop);
    err = hipGetLastError();



    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    //for (int i = 0; i < numElements; ++i)
    //{
    //    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    //    {
    //        fprintf(stderr, "Result verification failed at element %d!\n", i);
    //        exit(EXIT_FAILURE);
    //    }
    //}

    int idx1= 6;

    printf("Test PASSED\n");
    printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, __half2float(h_A[idx1].x), __half2float(h_A[idx1].y),
        __half2float(h_B[idx1].x), __half2float(h_B[idx1].y), __half2float(h_C[idx1].x), __half2float(h_C[idx1].y));

    printf("Kernel time: %f ms\n", milliseconds);

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

