#include "hip/hip_runtime.h"

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include "hip/hip_complex.h"
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const hipComplex *A, const hipComplex *B, hipComplex *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i].x = A[i].x + B[i].x;
        C[i].y = A[i].y + B[i].y;
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(hipComplex);
    printf("[Vector addition of %d elements]\n", numElements);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the host input vector A
    hipComplex *h_A = (hipComplex*)malloc(size);

    // Allocate the host input vector B
    hipComplex *h_B = (hipComplex*)malloc(size);

    // Allocate the host output vector C
    hipComplex *h_C = (hipComplex*)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i].x = rand()/(float)RAND_MAX;
        h_A[i].y = rand() / (float)RAND_MAX;
        h_B[i].x = rand()/(float)RAND_MAX;
        h_B[i].y = rand() / (float)RAND_MAX;
    }

    // Allocate the device input vector A
    hipComplex *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    hipComplex *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    hipComplex *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    hipEventRecord(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipEventRecord(stop);
    err = hipGetLastError();



    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    //for (int i = 0; i < numElements; ++i)
    //{
    //    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    //    {
    //        fprintf(stderr, "Result verification failed at element %d!\n", i);
    //        exit(EXIT_FAILURE);
    //    }
    //}

    int idx1;
    idx1 = 6;

    printf("Test PASSED\n");
    printf("Sample output on index %d: (%f+%fi)+(%f+%fi)=%f+%fi\n", idx1, h_A[idx1].x, h_A[idx1].y, h_B[idx1].x, h_B[idx1].y, h_C[idx1].x, h_C[idx1].y);

    printf("Kernel time: %f ms\n", milliseconds);

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

